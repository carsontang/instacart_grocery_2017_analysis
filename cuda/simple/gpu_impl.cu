#include "hip/hip_runtime.h"
#include "pybind_simple.h"

__global__ void kernel_add(int *i, int *j, int *result) {
    *result = *i + *j;
}

__global__ void kernel_add_arrays(int *arr1, int *arr2, int *result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] = arr1[idx] + arr2[idx];
    }
}

int add(int i, int j) {
    size_t nBytes = sizeof(i);
    int result = 0;

    int *d_i, *d_j, *d_result;
    hipMalloc( (int **)&d_i, nBytes);
    hipMalloc( (int **)&d_j, nBytes);
    hipMalloc( (int **)&d_result, nBytes);

    hipMemcpy(d_i, &i, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_j, &j, nBytes, hipMemcpyHostToDevice);

    kernel_add<<<1, 1>>>(d_i, d_j, d_result);

    hipMemcpy(&result, d_result, nBytes, hipMemcpyDeviceToHost);

    hipFree(d_i);
    hipFree(d_j);
    hipFree(d_result);

    hipDeviceReset();

    return result;
}

void parallel_add(int* arr1, int* arr2, int* result, int n) {
    size_t nBytes = n * sizeof(int);
    int *d_arr1, *d_arr2, *d_result;

    hipMalloc( (int **)&d_arr1, nBytes);
    hipMalloc( (int **)&d_arr2, nBytes);
    hipMalloc( (int **)&d_result, nBytes);

    hipMemcpy(d_arr1, arr1, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_arr2, arr2, nBytes, hipMemcpyHostToDevice);

    kernel_add_arrays<<<32, 32>>>(d_arr1, d_arr2, d_result, n);

    hipMemcpy(result, d_result, nBytes, hipMemcpyDeviceToHost);

    hipFree(d_arr1);
    hipFree(d_arr2);
    hipFree(d_result);

    hipDeviceReset();
}