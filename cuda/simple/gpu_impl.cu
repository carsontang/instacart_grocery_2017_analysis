#include "hip/hip_runtime.h"
#include "pybind_simple.h"

__global__ void kernel_add(int *i, int *j, int *result) {
    *result = *i + *j;
}

int add(int i, int j) {
    size_t nBytes = sizeof(i);
    int result = 0;

    int *d_i, *d_j, *d_result;
    hipMalloc( (int **)&d_i, nBytes);
    hipMalloc( (int **)&d_j, nBytes);
    hipMalloc( (int **)&d_result, nBytes);

    hipMemcpy(d_i, &i, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_j, &j, nBytes, hipMemcpyHostToDevice);

    kernel_add<<<1, 1>>>(d_i, d_j, d_result);

    hipMemcpy(&result, d_result, hipMemcpyDeviceToHost);

    hipFree(d_i);
    hipFree(d_j);
    hipFree(d_result);

    return result;
}